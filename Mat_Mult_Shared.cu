#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#define Tile_size 2
// This code assumes that your device support block size of 1024
#define MAX_RANGE 9

#define funcCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            printf( "Failed to run stmt %d ", __LINE__);                       \
            printf( "Got CUDA error ...  %s ", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)

// Compute C = A * B
__global__ void matrixMultiplyShared(float * A, float * B, float * C,
                                    int numARows, int numAColumns,
                                    int numBRows, int numBColumns,
                                    int numCRows, int numCColumns)
{
    __shared__ float sA[Tile_size][Tile_size];   // Tile size
    __shared__ float sB[Tile_size][Tile_size];

    int Row = blockDim.y*blockIdx.y + threadIdx.y;
    int Col = blockDim.x*blockIdx.x + threadIdx.x;
    float Cvalue = 0.0;
    sA[threadIdx.y][threadIdx.x] = 0.0;
    sB[threadIdx.y][threadIdx.x] = 0.0;

    for (int k = 0; k < (((numAColumns - 1)/ Tile_size) + 1); k++)
    {
        if ( (Row < numARows) && (threadIdx.x + (k*Tile_size)) < numAColumns)
        {
            sA[threadIdx.y][threadIdx.x] = A[(Row*numAColumns) + threadIdx.x + (k*Tile_size)];
        }
        else
        {
            sA[threadIdx.y][threadIdx.x] = 0.0;
        }
        if ( Col < numBColumns && (threadIdx.y + k*Tile_size) < numBRows)
        {
            sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + k*Tile_size)*numBColumns + Col];
        }
        else
        {
            sB[threadIdx.y][threadIdx.x] = 0.0;
        }
        __syncthreads();

        for (int j = 0; j < Tile_size; ++j)
        {
            Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
        }
    }
    if (Row < numCRows && Col < numCColumns)
    {
        C[Row*numCColumns + Col] = Cvalue;
    }
}

void Print_Mat(int Row,int Col,float * Mat)
{
	for(int i=0;i<Row*Col;i++)
			{
			printf("%f  ",*(Mat+i));

			if((i%Col)==0 )
				{
					printf("\n");
				}
			}
}//Function close


void matMultiplyOnHost(float * A, float * B, float * C, int numARows,
                        int numAColumns, int numBRows, int numBColumns,
                        int numCRows, int numCColumns)
{
    for (int i=0; i < numARows; i ++)
    {
        for (int j = 0; j < numAColumns; j++)
        {
            C[i*numCColumns + j ] = 0.0;
            for (int k = 0; k < numCColumns; k++)
            {
                C[i*numCColumns + j ] += A[i*numAColumns + k] * B [k*numBColumns + j];
            }
        }
    }
    return;
}

int main(int argc, char ** argv) {
    float * hostA; // The A matrix
    float * hostB; // The B matrix
    float * hostC; // The output C matrix
    float * hostComputedC;
    float * deviceA;
    float * deviceB;
    float * deviceC;

    // Please adjust rows and columns according to you need.
    int numARows = 4; // number of rows in the matrix A
    int numAColumns = 4; // number of columns in the matrix A
    int numBRows = 4; // number of rows in the matrix B
    int numBColumns = 4; // number of columns in the matrix B

    int numCRows; // number of rows in the matrix C (you have to set this)
    int numCColumns; // number of columns in the matrix C (you have to set this)

    hostA = (float *) malloc(sizeof(float)*numARows*numAColumns);
    hostB = (float *) malloc(sizeof(float)*numBRows*numBColumns);

    for (int i = 0; i < numARows*numAColumns; i++)
    {
        //hostA[i] = (rand() % MAX_RANGE) / 2.0;
    	hostA[i]=1.0;
    }
    for (int i = 0; i < numBRows*numBColumns; i++)
    {
        //hostB[i] = (rand() % MAX_RANGE) / 2.0;
    	hostB[i]=1.0;
    }

    printf("\nMatrix A Values:\n");
    Print_Mat(numARows,numAColumns,hostA);//Function Call

    printf("\n\nMatrix B Values:\n");
    Print_Mat(numBRows,numBColumns,hostB);//Function Call



    // Setting numCRows and numCColumns
    numCRows = numARows;
    numCColumns = numBColumns;

    hostC = (float *) malloc(sizeof(float)*numCRows*numCColumns);
    hostComputedC = (float *) malloc(sizeof(float)*numCRows*numCColumns);

    // Allocating GPU memory
    funcCheck(hipMalloc((void **)&deviceA, sizeof(float)*numARows*numAColumns));
    funcCheck(hipMalloc((void **)&deviceB, sizeof(float)*numBRows*numBColumns));
    funcCheck(hipMalloc((void **)&deviceC, sizeof(float)*numCRows*numCColumns));

    // Copy memory to the GPU
    funcCheck(hipMemcpy(deviceA, hostA, sizeof(float)*numARows*numAColumns, hipMemcpyHostToDevice));
    funcCheck(hipMemcpy(deviceB, hostB, sizeof(float)*numBRows*numBColumns, hipMemcpyHostToDevice));

    // Initialize the grid and block dimensions
    dim3 dimBlock(Tile_size, Tile_size, 1);
    dim3 dimGrid((numCColumns/Tile_size) + 1, (numCRows/Tile_size) + 1, 1);

    //@@ Launch the GPU Kernel here
    matrixMultiplyShared<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

    hipError_t err1 = hipPeekAtLastError();
    hipDeviceSynchronize();
    printf( "Got CUDA error ... %s \n", hipGetErrorString(err1));

    // Copy the results in GPU memory back to the CPU
    funcCheck(hipMemcpy(hostC, deviceC, sizeof(float)*numCRows*numCColumns, hipMemcpyDeviceToHost));

    printf("\nMatrix C From Device\n");
    Print_Mat(numCRows,numCColumns,hostC);//Function Call

    matMultiplyOnHost(hostA, hostB, hostComputedC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

    printf("\nMatrix C From Host\n");
    Print_Mat(numCRows,numCColumns,hostComputedC);//Function Call

    for (int i=0; i < numCColumns*numCRows; i++)
    {
        if (hostComputedC[i]  != hostC[i] )
        {
            printf("Mismatch at Row = %d Col = %d hostComputed[] = %f --device[] %f\n", i / numCColumns, i % numCColumns, hostComputedC[i], hostC[i]);
            break;
        }
    }
    // Free the GPU memory
    funcCheck(hipFree(deviceA));
    funcCheck(hipFree(deviceB));
    funcCheck(hipFree(deviceC));

    free(hostA);
    free(hostB);
    free(hostC);
    free(hostComputedC);

    return 0;
}
